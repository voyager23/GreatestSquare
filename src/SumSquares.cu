/*
 * SumSquares.cu
 * 
 * Copyright 2021 mike <mike@fedora33>
 * 
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 * 
 * 
 */


#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/driver_types.h>	// cudaError_t
#include <hip/hip_vector_types.h>
	// cudaMalloc, cudaFree

__global__ void kernel(ulong* d_squares, const ulong n_squares, ulong* d_results, ulong N) {
	ulong i = threadIdx.x + (blockIdx.x * blockDim.x);
	if(i < N) {
		// scan in reverse the squares array
		// save first square which divides i in results[i]
		if(i == 0) d_squares[i] = 0;
		if(i < 4) d_squares[i] = 1;
		for(int x = n_squares-1; x > 0; x -= 1) {
			if((i % d_squares[x]) == 0) {
				d_results[i] = x*x;
				break;
			}
		}
	}
}

int main(int argc, char **argv)
{
	hipError_t error_id;
	
	// Allocate and set the host 'squares' array
	ulong N = 128;	
	ulong root_max = (ulong)floor(sqrt((double)N));	
	const ulong n_squares = root_max + 1;	
	ulong h_squares[n_squares];
	for(int x = 0; x < n_squares; x += 1) h_squares[x] = x*x;
	
	// Allocate host results array
	ulong h_results[N];
	
	// Allocate memory on device for 'squares'
	ulong *d_squares;
	error_id = hipMalloc((void**)&d_squares, sizeof(ulong)*n_squares);
	if(error_id != hipSuccess) {
		printf("hipMalloc squares failed with %d\n", error_id);
		exit(1);
	}	
	// Copy squares to device
	error_id = hipMemcpy(d_squares, h_squares, sizeof(ulong)*n_squares,
		hipMemcpyHostToDevice);
	if(error_id != hipSuccess) {
		printf("hipMemcpy squares to device failed with %d\n", error_id);
		exit(1);
	}
	// Allocate memory on device for N results
	ulong *d_results;
	error_id = hipMalloc((void**)&d_results, sizeof(ulong)*N);
	if(error_id != hipSuccess) {
		printf("hipMalloc results failed with %d\n", error_id);
		exit(1);
	}
	
	// Set configuration parameters
	dim3 grid_size=(1); dim3 block_size=(N);
	
	// launch kernel
	kernel<<<grid_size, block_size>>>(d_squares, n_squares, d_results, N);
	
	// Wait for device to finish?
	//cudaDeviceSynchronize();
	
	// copy N results back to host
	error_id = hipMemcpy(h_results, d_results, sizeof(ulong)*N,
		hipMemcpyDeviceToHost);
	if(error_id != hipSuccess) {
		printf("hipMemcpy to host  failed with %d\n", error_id);
		exit(1);
	}
	
	// Print results array
	for(int x = 0; x < N; ++x) printf("%d:%ld  ", x, h_results[x]);
	printf("\n");

	// Cleanup
	hipFree(d_squares);
	hipFree(d_results);
	
	return 0;
}

