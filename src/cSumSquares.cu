/*
 * cSumSquares.cu
 * 
 * Copyright 2021 mike <mike@fedora33>
 * 
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 * 
 * =====================================================================
 * Function g(n) is defined as greatest perfect square which divides n.
 * Consider n = 10^9. 31623^2 = 1000014129
 * To build a table of useful perfect squares: for x in range(1, 31624) calc x^2
 * Requires 976577 blocks of 1024 threads
 * Host memory approx 14Gb free, 10^9
 * 
 * =====================================================================
 */

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define DEBUG 1
#define MANAGED 0

__global__ void set_squares(long *d_squares, long n_squares) {
	long i = threadIdx.x + (blockIdx.x * blockDim.x);
	if(i < n_squares) d_squares[i] = (int)(i+1)*(i+1);
}

//__global__ void func_g(int* d_squares, const long limit, long *h_sums, long N) {
__global__ void func_g() {
	

	return;
	//END DEBUG
#if(0)	
	long i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i < N) {
		// scan in reverse the squares array
		// save first square which divides i in results[i]
		if(i > 3) {
			for(long x = limit-1; x > 0; x -= 1) {
				if((i % d_squares[x]) == 0) {
					h_sums[i] = d_squares[x];
					break;
				}
			} // for...
		} else {
			h_sums[i] = i;
		}
	} //
#endif
}

int main(int argc, char **argv)
{
	hipError_t error_id;
	long *d_squares = NULL;
	long *h_sums = NULL;
	
	// extract target N
	long x = 0;
	if(argc == 2) {
		x = atol(argv[1]);
	} else {
		printf("usage: css target (< 1e8)\n");
		exit(1);
	}	
	const long N = x;
	if(N <= 1e9L) {
		printf("target: %ld\n", N);
	} else {
		printf("target: %ld exceeds program limitations\n", N);
		exit(2);
	}
	// determine array dimensions
	long limit = (long)(sqrt(sqrt(N)) + 1);	// defines size of array	

#if(DEBUG)
		printf("target: %ld		limit: %ld\n", N, limit);
#endif

	// Allocate space on device
	error_id = hipMalloc(&d_squares, sizeof(long )*limit);
	if(error_id != hipSuccess) {
		printf("hipMalloc squares failed with %d\n", error_id);
		exit(1);
	}	
	// launch the generator on kernel
	set_squares<<<1,limit>>>(d_squares, limit);
	hipDeviceSynchronize();

#if(DEBUG)	
		// allocate space on host and copy device squares
		long *h_squares = (long *)malloc(sizeof(long )*limit);
		hipMemcpy(h_squares, d_squares, sizeof(long )*limit, hipMemcpyDeviceToHost);
		// prlong array
		for(long x = 0; x < limit; ++x) printf("%d:%ld  ", x, h_squares[x]); printf("\n");
		// clear host array
		free(h_squares);
#endif

#if(MANAGED)
	// Allocate managed memory for results
	// Note: Fails for more than 1e8 long int
	error_id = cudaMallocManaged(&h_sums, sizeof(long)*N);
	if(error_id != cudaSuccess) {
		printf("cudaMallocManaged sums failed with %d\n", error_id);
		exit(1);
	}
#else
	// allocate host memory (h_sums)
	h_sums = (long*)malloc(sizeof(long)*N);
	if(h_sums == NULL){
		printf("malloc() h_sums failed.\n");
		exit(1);
	}
	// allocate device memory (d_sums)
	
#endif
	
	// calculate the launch config based in thread blocks of 1024 threads
	int nBlocks = (N/1024) + 1;
	// LIMIT OF nBlocks = 2147483647; // DEBUG kernel returns in 41s
	// Value of INT_MAX is +2147483647 (32 bits)
	// Using 1e9 as program limit then 976562 blocks are required
	// launch the kernel
	func_g<<<nBlocks,1024>>>();
	error_id = hipDeviceSynchronize();
	if(error_id != hipSuccess) {
		printf("Kernel launch returned error code %d\n", error_id);
		exit(1);
	} else {
		printf("\nkernel launched with %d blocks\n", nBlocks);
	}
	
	// code to calc total sum of h_sums
	// if NOT using ManagedMemory copy device to host
	if(!MANAGED)
	
	// output S(N): total 
	
	// CleanUp
	hipFree(d_squares);
	
#if(MANAGED)
	cudaFree(h_sums);
#else
	free(h_sums);
#endif
	return 0;
}

